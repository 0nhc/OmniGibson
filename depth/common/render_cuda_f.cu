#include <cstdlib>
//#include <cstdio>
#include <stdio.h>
#include <vector>
#include <algorithm>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

const int N_THREADS = 64;
const int N_BLOCKS = 64;

__global__ void copy_mem(unsigned char *source, unsigned char *render)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
      for (int channel = 0; channel < 3; channel ++ )
        render[3*((y+j)*width + x) + channel] = source[3 * ((y+j)*width + x) + channel];
}


__global__ void set_depth(unsigned int *depth)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
      depth[(y+j)*width + x] = 65535;
}


__global__ void char_to_int(int * img2, unsigned char * img)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
      img2[(y+j)*width + x] =  img[3*((y+j)*width + x) + 0] * 256 * 256 + img[3*((y+j)*width + x) + 1] * 256 + img[3*((y+j)*width + x) + 2];
}


__global__ void int_to_char(int * img2, unsigned char * img)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS) {
      img[3*((y+j)*width + x)] = img2[(y+j)*width + x] / (256*256);
      img[3*((y+j)*width + x)+1] = img2[(y+j)*width + x] / 256 % 256;
      img[3*((y+j)*width + x)+2] = img2[(y+j)*width + x] % 256;
    }
}


__global__ void to3d_point(float *depth, float *points3d)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  int h = w / 2;
    
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     
     int iw = x;
     int ih = y + j;
     float depth_point = depth[ih*w + iw] * 128.0;
     float phi = ((float)(ih) + 0.5) / float(h) * M_PI;
     float theta = ((float)(iw) + 0.5) / float(w) * 2 * M_PI + M_PI;
  
      points3d[(ih * w + iw) * 4 + 0] = depth_point * sin(phi) * cos(theta);
      points3d[(ih * w + iw) * 4 + 1] = depth_point * sin(phi) * sin(theta);
      points3d[(ih * w + iw) * 4 + 2] = depth_point * cos(phi);
      points3d[(ih * w + iw) * 4 + 3] = 1;
  
  }
}

__global__ void transform(float *points3d_after, float *points3d, float * transformation_matrix)
{
 int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     int iw = x;
     int ih = y + j;
     for (int ic = 0; ic < 3; ic ++) {
     points3d_after[(ih * w + iw) * 3 + ic] = points3d[(ih * w + iw) * 4 + 0] * transformation_matrix[4 * ic + 0]
     + points3d[(ih * w + iw) * 4 + 1] * transformation_matrix[4 * ic + 1] 
     + points3d[(ih * w + iw) * 4 + 2] * transformation_matrix[4 * ic + 2] 
     + points3d[(ih * w + iw) * 4 + 3] * transformation_matrix[4 * ic + 3]; 
    }
  }
}


__global__ void transform2d(float *points3d_after, float *points3d_polar)
{
 int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     int iw = x;
     int ih = y + j;
     float x = points3d_after[(ih * w + iw) * 3 + 0];
     float y = points3d_after[(ih * w + iw) * 3 + 1];
     float z = points3d_after[(ih * w + iw) * 3 + 2];

    points3d_polar[(ih * w + iw) * 3 + 0] = sqrt(x * x + y * y + z * z);
    points3d_polar[(ih * w + iw) * 3 + 1] = atan2(y, x);
    points3d_polar[(ih * w + iw) * 3 + 2] = atan2(sqrt(x * x + y * y), z);
  }
}


__global__ void render_depth(float *points3d_polar, unsigned int * depth_render)
{
 int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  int h = w /2;
  
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     int iw = x;
     int ih = y + j;
     int tx = round((points3d_polar[(ih * w + iw) * 3 + 1] + M_PI)/(2*M_PI) * w - 0.5);
     int ty = round((points3d_polar[(ih * w + iw) * 3 + 2])/M_PI * h - 0.5);
     int this_depth = (int)(512 * points3d_polar[(ih * w + iw) * 3 + 0]);
     atomicMin(&depth_render[(ty * w + tx)] , this_depth);
  }
}



__global__ void render_final(float *points3d_polar, int * depth_render, int * img,  int * render)
{
 int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  int h = w /2;
  
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     int iw = x;
     int ih = y + j;
     int tx = round((points3d_polar[(ih * w + iw) * 3 + 1] + M_PI)/(2*M_PI) * w - 0.5);
     int ty = round((points3d_polar[(ih * w + iw) * 3 + 2])/M_PI * h - 0.5);
     int this_depth = (int)(12800/128 * points3d_polar[(ih * w + iw) * 3 + 0]);
     int delta = this_depth - depth_render[(ty * w + tx)];
     
     //printf("%d %d\n", this_depth, depth_render[(ty * w + tx)]);
     if ((y > h/8) && (y < h*7/8))
     if ((delta > -10) && (delta < 10) && (this_depth < 10000)) {
           render[(ty * w + tx)] = img[(ih * w + iw)];
     }
  }
}


__global__ void transform_cube_to_equi(float *dst, float * src, uint * idxs,  size_t count)
{
  int n_to_do = count / ( gridDim.x* blockDim.x);
  int start = (blockIdx.x * blockDim.x + threadIdx.x) * n_to_do;
  //printf("x: %d w: %d | %d %d (%d)(%d)\n", blockIdx.x, threadIdx.x, gridDim.x, blockDim.x, start, n_to_do);
  for (int j = 0; j < n_to_do; j++)
  {
    dst[start + j] = src[idxs[start + j]];
  }
}

__global__ void blue(float * dst, float * src, size_t count)
{
  int n_to_do = count / ( gridDim.x* blockDim.x);
  int start = (blockIdx.x * blockDim.x + threadIdx.x) * n_to_do;
  printf("%d", n_to_do);
  for (int j = 0; j < n_to_do; j++)
  {
    dst[start + j] = src[3*(start + j) + 2];
  }
}


__global__ void readSurfaceToCubeMapBuffer(float * dst, hipSurfaceObject_t surf2D, int width, int height)
{
    unsigned int start_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int start_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int n_to_do_x = height / (blockDim.y * gridDim.y);
    unsigned int n_to_do_y = width / (blockDim.x * gridDim.x);
    unsigned int n_to_do = n_to_do_x * n_to_do_y;
    int start = (blockIdx.x * blockDim.x + threadIdx.x) * n_to_do;
    float temp;
    for (int j = 0; j < n_to_do; j++)
    {
        int x_val = (start_x + (j/n_to_do_y));//*4;
        int y_val = start_y + (j%n_to_do_y);
        surf2Dread(&temp, surf2D, x_val, y_val );
        dst[start + j] = temp;
        //src[3*(start + j) + 2];
    }
    printf("Thread index: (%i, %i); hipArray = %d\n", n_to_do_x, n_to_do_y , n_to_do);
}

extern "C"{

void fillBlue(float * dst, hipArray_t src, size_t offset)
{
    hipResourceDesc wdsc;
    wdsc.resType = hipResourceTypeArray;
    wdsc.res.array.array = src;
    hipSurfaceObject_t writeSurface;
    hipCreateSurfaceObject(&writeSurface, &wdsc);
    dim3 dimBlock(1, 1);
    dim3 dimGrid(1, 1);
    printf("%d\n", sizeof(src));
    readSurfaceToCubeMapBuffer<<< dimGrid, dimBlock >>>(dst + offset, writeSurface, 768, 768);
}

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

uint* move_idxs_to_gpu(uint * cube_idx_to_equi, size_t count) {
    uint *d_idx;
    const int idxs_mem_size = count*sizeof(uint);
    hipMalloc((void **)&d_idx, idxs_mem_size);
    hipMemcpy(d_idx, cube_idx_to_equi, idxs_mem_size, hipMemcpyHostToDevice);
    return d_idx;
}

float* allocate_buffer_on_gpu(size_t count) {
    float *d_dst;
    const int dst_mem_size = count*sizeof(float);
    hipMalloc((void **)&d_dst, dst_mem_size);
    return d_dst;
}

void render(int h,int w,unsigned char * img, float * depth,float * pose, unsigned char * render, int * depth_render){
    //int ih, iw, i, ic;
    
    const int nx = w;
    const int ny = h;
    const int depth_mem_size = nx*ny*sizeof(float);
    const int frame_mem_size = nx*ny*sizeof(unsigned char) * 3;
    
    dim3 dimGrid(nx/TILE_DIM, ny/TILE_DIM, 1);
    dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
    
    unsigned char *d_img, *d_render;
    float *d_depth, *d_pose;
    int *d_depth_render;
    float *d_3dpoint, *d_3dpoint_after, *d_3dpoint_polar;
    
    int *d_render2, *d_img2;
    
    hipMalloc((void **)&d_img, frame_mem_size);
    hipMalloc((void **)&d_render, frame_mem_size);
    hipMalloc((void **)&d_depth, depth_mem_size);
    hipMalloc((void **)&d_depth_render, nx * ny * sizeof(int));
    hipMalloc((void **)&d_3dpoint, depth_mem_size * 4);
    hipMalloc((void **)&d_3dpoint_after, depth_mem_size * 4);
    hipMalloc((void **)&d_3dpoint_polar, depth_mem_size * 4);
    hipMalloc((void **)&d_pose, sizeof(float) * 16);
    hipMalloc((void **)&d_render2, nx * ny * sizeof(int));
    hipMalloc((void **)&d_img2, nx * ny * sizeof(int));
    
    hipMemcpy(d_depth_render, depth_render, nx * ny * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_pose, pose, sizeof(float) * 16, hipMemcpyHostToDevice);
    hipMemcpy(d_img, img, frame_mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_depth, depth, depth_mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_render, render, frame_mem_size, hipMemcpyHostToDevice);
    
    hipMemset(d_render2, 0, nx * ny * sizeof(int));
    hipMemset(d_img2, 0, nx * ny * sizeof(int));
    
    hipMemset(d_3dpoint, 0, depth_mem_size * 4);
    hipMemset(d_3dpoint_after, 0, depth_mem_size * 4);
    
    to3d_point<<< dimGrid, dimBlock >>>(d_depth, d_3dpoint);
    transform<<< dimGrid, dimBlock >>>(d_3dpoint_after, d_3dpoint, d_pose);
    transform2d<<<dimGrid, dimBlock>>>(d_3dpoint_after, d_3dpoint_polar);
    
    char_to_int <<< dimGrid, dimBlock >>> (d_img2, d_img);
    char_to_int <<< dimGrid, dimBlock >>> (d_render2, d_render);
    
    //render_depth <<< dimGrid, dimBlock >>> (d_3dpoint_polar, d_depth_render);
    render_final <<< dimGrid, dimBlock >>> (d_3dpoint_polar, d_depth_render, d_img2, d_render2);
    
    int_to_char <<< dimGrid, dimBlock >>> (d_render2, d_render);
    
    hipMemcpy(render, d_render, frame_mem_size, hipMemcpyDeviceToHost);
    //cudaMemcpy(depth_render, d_depth_render, nx * ny * sizeof(unsigned int), cudaMemcpyDeviceToHost);
        
    hipFree(d_img);
    hipFree(d_depth);
    hipFree(d_render2);
    hipFree(d_img2);
    hipFree(d_render);
    hipFree(d_depth_render);
    hipFree(d_3dpoint);
    hipFree(d_3dpoint_after);
    hipFree(d_3dpoint_polar);
    hipFree(d_pose);
}


void cube_to_equi(float * dst, float * d_src, uint *d_idx, size_t count, size_t src_size){
    // First call move_idxs_to_gpu!

    const int dst_mem_size = count*sizeof(float);
    const int src_mem_size = src_size*sizeof(float);
    //const int idxs_mem_size = count*sizeof(uint);
    float *d_dst;//, *d_src;

    hipMalloc((void **)&d_dst, dst_mem_size);
    //cudaMalloc((void **)&d_src, src_mem_size);
    //cudaMalloc((void **)&d_idx, idxs_mem_size);
    
    hipMemcpy(d_dst, dst, dst_mem_size, hipMemcpyHostToDevice);
    //cudaMemcpy(d_src, src, src_mem_size, cudaMemcpyHostToDevice);
    //cudaMemcpy(d_idx, cube_idx_to_equi, idxs_mem_size, cudaMemcpyHostToDevice);
    
    transform_cube_to_equi<<< N_BLOCKS, N_THREADS >>>(d_dst, d_src, d_idx, count);
    
    //cudaMemset(d_dst, 0, dst_mem_size);
    hipMemcpy(dst, d_dst, dst_mem_size, hipMemcpyDeviceToHost);

    hipFree(d_dst);
    //cudaFree(d_src);
    //cudaFree(d_idx);
    hipDeviceSynchronize();
}

}//extern "C"