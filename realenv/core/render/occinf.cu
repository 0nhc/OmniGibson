#include <cstdlib>
#include <cstdio>
#include <vector>
#include <algorithm>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

__global__ void copy_mem(unsigned char *source, unsigned char *render)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
      for (int channel = 0; channel < 3; channel ++ )
        render[3*((y+j)*width + x) + channel] = source[3 * ((y+j)*width + x) + channel];
}


__global__ void set_depth(unsigned int *depth)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
      depth[(y+j)*width + x] = 65535;
}


__global__ void char_to_int(int * img2, unsigned char * img)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
      img2[(y+j)*width + x] =  img[3*((y+j)*width + x) + 0] * 256 * 256 + img[3*((y+j)*width + x) + 1] * 256 + img[3*((y+j)*width + x) + 2];
}


__global__ void int_to_char(int * img2, unsigned char * img)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS) {
      img[3*((y+j)*width + x)] = img2[(y+j)*width + x] / (256*256);
      img[3*((y+j)*width + x)+1] = img2[(y+j)*width + x] / 256 % 256;
      img[3*((y+j)*width + x)+2] = img2[(y+j)*width + x] % 256;
    }
}


__global__ void to3d_point(float *depth, float *points3d)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  int h = w / 2;
    
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     
     int iw = x;
     int ih = y + j;
     float depth_point = depth[ih*w + iw] * 128.0;
     float phi = ((float)(ih) + 0.5) / float(h) * M_PI;
     float theta = ((float)(iw) + 0.5) / float(w) * 2 * M_PI + M_PI;
  
      points3d[(ih * w + iw) * 4 + 0] = depth_point * sin(phi) * cos(theta);
      points3d[(ih * w + iw) * 4 + 1] = depth_point * sin(phi) * sin(theta);
      points3d[(ih * w + iw) * 4 + 2] = depth_point * cos(phi);
      points3d[(ih * w + iw) * 4 + 3] = 1;
  
  }
}

__global__ void transform(float *points3d_after, float *points3d, float * transformation_matrix)
{
 int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     int iw = x;
     int ih = y + j;
     for (int ic = 0; ic < 3; ic ++) {
     points3d_after[(ih * w + iw) * 3 + ic] = points3d[(ih * w + iw) * 4 + 0] * transformation_matrix[4 * ic + 0]
     + points3d[(ih * w + iw) * 4 + 1] * transformation_matrix[4 * ic + 1] 
     + points3d[(ih * w + iw) * 4 + 2] * transformation_matrix[4 * ic + 2] 
     + points3d[(ih * w + iw) * 4 + 3] * transformation_matrix[4 * ic + 3]; 
    }
  }
}


__global__ void transform2d(float *points3d_after, float *points3d_polar)
{
 int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     int iw = x;
     int ih = y + j;
     float x = points3d_after[(ih * w + iw) * 3 + 0];
     float y = points3d_after[(ih * w + iw) * 3 + 1];
     float z = points3d_after[(ih * w + iw) * 3 + 2];

    points3d_polar[(ih * w + iw) * 3 + 0] = sqrt(x * x + y * y + z * z);
    points3d_polar[(ih * w + iw) * 3 + 1] = atan2(y, x);
    points3d_polar[(ih * w + iw) * 3 + 2] = atan2(sqrt(x * x + y * y), z);
  }
}


__global__ void render_depth(float *points3d_polar, unsigned int * depth_render)
{
 int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  int h = w /2;
  
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     int iw = x;
     int ih = y + j;
     int tx = round((points3d_polar[(ih * w + iw) * 3 + 1] + M_PI)/(2*M_PI) * w - 0.5);
     int ty = round((points3d_polar[(ih * w + iw) * 3 + 2])/M_PI * h - 0.5);
     int this_depth = (int)(100 * points3d_polar[(ih * w + iw) * 3 + 0]);
     atomicMin(&depth_render[(ty * w + tx)] , this_depth);
  }
}




__global__ void render_occu(float *points3d_polar, unsigned int * depth_render, bool * occu_map, float * depth)
{
 int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int w = gridDim.x * TILE_DIM;
  int h = w /2;
  
  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
  {
     int iw = x;
     int ih = y + j;
     int tx = round((points3d_polar[(ih * w + iw) * 3 + 1] + M_PI)/(2*M_PI) * w - 0.5);
     int ty = round((points3d_polar[(ih * w + iw) * 3 + 2])/M_PI * h - 0.5);
     
     float depth_point = depth[ih*w + iw] * 100.0 * 128;
     
     int this_depth = (int)(100 * points3d_polar[(ih * w + iw) * 3 + 0]);
     if ((this_depth - depth_render[(ty * w + tx)]) > 5) {
     
         int scale = int(depth_point / float(this_depth) * 4);
         
         for (int j = -scale; j < scale; j ++)
             for (int k = -scale; k < scale; k++)
                 {
                 if ((ty + j < h) && (ty + j > 0) && (tx + k > 0) && (tx + k < w))
                     occu_map[((ty+j) * w + (tx+k))] = 1;
                 };
     }
  }
}




extern "C"{
    
void occinf(int h,int w, float * depth,float * pose, bool * occmap, unsigned int * depth_render){
    const int nx = w;
    const int ny = h;
    const int depth_mem_size = nx*ny*sizeof(float);
    const int occ_map_size = nx*ny*sizeof(bool);
    
    
    dim3 dimGrid(nx/TILE_DIM, ny/TILE_DIM, 1);
    dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
    
    float *d_depth, *d_pose;
    unsigned int *d_depth_render;
    float *d_3dpoint, *d_3dpoint_after, *d_3dpoint_polar;
    bool *d_occu;
    
    hipMalloc((void **)&d_depth, depth_mem_size);
    hipMalloc((void **)&d_depth_render, nx * ny * sizeof(unsigned int));
    hipMalloc((void **)&d_3dpoint, depth_mem_size * 4);
    hipMalloc((void **)&d_3dpoint_after, depth_mem_size * 4);
    hipMalloc((void **)&d_3dpoint_polar, depth_mem_size * 4);
    hipMalloc((void **)&d_pose, sizeof(float) * 16);
    hipMalloc((void **)&d_occu, occ_map_size);

    
    hipMemcpy(d_depth_render, depth_render, nx * ny * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_pose, pose, sizeof(float) * 16, hipMemcpyHostToDevice);
    hipMemcpy(d_depth, depth, depth_mem_size, hipMemcpyHostToDevice);
    
    hipMemset(d_3dpoint, 0, depth_mem_size * 4);
    hipMemset(d_3dpoint_after, 0, depth_mem_size * 4);
    hipMemset(d_occu, 0, occ_map_size);
    
    
    to3d_point<<< dimGrid, dimBlock >>>(d_depth, d_3dpoint);
    transform<<< dimGrid, dimBlock >>>(d_3dpoint_after, d_3dpoint, d_pose);
    transform2d<<<dimGrid, dimBlock>>>(d_3dpoint_after, d_3dpoint_polar);
        
    render_depth <<< dimGrid, dimBlock >>> (d_3dpoint_polar, d_depth_render);
    render_occu <<< dimGrid, dimBlock >>> (d_3dpoint_polar, d_depth_render, d_occu, d_depth);
    hipMemcpy(occmap, d_occu, occ_map_size, hipMemcpyDeviceToHost);
    
    
    hipFree(d_occu);
    hipFree(d_depth);
    hipFree(d_depth_render);
    hipFree(d_3dpoint);
    hipFree(d_3dpoint_after);
    hipFree(d_3dpoint_polar);
    hipFree(d_pose);
}
    
    
}//extern "C"